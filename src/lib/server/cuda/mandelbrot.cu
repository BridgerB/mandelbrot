#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define WIDTH 4096
#define HEIGHT 4096
#define MAX_ITER 1000
#define X_MIN -2.0
#define X_MAX 1.0
#define Y_MIN -1.5
#define Y_MAX 1.5

__global__ void mandelbrot_kernel(int *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * WIDTH + idx;

    if (idx >= WIDTH || idy >= HEIGHT) return;

    double x0 = X_MIN + (X_MAX - X_MIN) * idx / (double)WIDTH;
    double y0 = Y_MIN + (Y_MAX - Y_MIN) * idy / (double)HEIGHT;
    double x = 0.0, y = 0.0;
    int iter = 0;

    while (x * x + y * y <= 4.0 && iter < MAX_ITER) {
        double xtemp = x * x - y * y + x0;
        y = 2.0 * x * y + y0;
        x = xtemp;
        iter++;
    }

    output[index] = iter;
}

// This function now writes the PPM data to the provided FILE stream.
void write_ppm(FILE *stream, int *data, int width, int height) {
    fprintf(stream, "P6\n%d %d\n255\n", width, height);
    for (int i = 0; i < width * height; i++) {
        unsigned char color = (data[i] == MAX_ITER) ? 0 : (data[i] % 255);
        fputc(color, stream); // R
        fputc(color, stream); // G
        fputc(color, stream); // B
    }
}

int main(int argc, char** argv) {
    int *d_output, *h_output;
    size_t size = WIDTH * HEIGHT * sizeof(int);

    // Allocate host and device memory
    h_output = (int*)malloc(size);
    hipMalloc(&d_output, size);

    // Set up grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((WIDTH + blockDim.x - 1) / blockDim.x, (HEIGHT + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    mandelbrot_kernel<<<gridDim, blockDim>>>(d_output);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA synchronization error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Copy result to host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    // Write to standard output
    write_ppm(stdout, h_output, WIDTH, HEIGHT);

    // Clean up
    free(h_output);
    hipFree(d_output);

    return 0;
}